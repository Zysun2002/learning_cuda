#include "hip/hip_runtime.h"
# include <torch/extension.h>
# include "utils.h"

template <typename scalar_t>
__global__ void trilinear_bw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> dL_dfeat_interp,
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> feats,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> points,
    torch:: PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> dL_dfeats
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int f = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (n > feats.size(0) || f > feats.size(2)) return;

    const scalar_t u = (points[n][0]+1)/2;
    const scalar_t v = (points[n][1]+1)/2;
    const scalar_t w = (points[n][2]+1)/2;

    const scalar_t a = (1-v)*(1-w);
    const scalar_t b = (1-v)*w;
    const scalar_t c = v*(1-w);
    const scalar_t d = 1-a-b-c;

    dL_dfeats[n][0][f] = (1-u)*a*dL_dfeat_interp[n][f];
    dL_dfeats[n][1][f] = (1-u)*b*dL_dfeat_interp[n][f];
    dL_dfeats[n][2][f] = (1-u)*c*dL_dfeat_interp[n][f];
    dL_dfeats[n][3][f] = (1-u)*d*dL_dfeat_interp[n][f];
    dL_dfeats[n][4][f] = u*a*dL_dfeat_interp[n][f];
    dL_dfeats[n][5][f] = u*b*dL_dfeat_interp[n][f];
    dL_dfeats[n][6][f] = u*c*dL_dfeat_interp[n][f];
    dL_dfeats[n][7][f] = u*d*dL_dfeat_interp[n][f];
}

//每个kernal的动作内容
template <typename scalar_t>
__global__ void trilinear_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> feats,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> points,
    torch:: PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> feat_interp
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int f = blockIdx.y * blockDim.y + threadIdx.y;
    // 取得每一个thread编号（n方向和f方向）
    
    //如果当前kernal不工作：
    if (n > feats.size(0) || f > feats.size(2)) return;

    //计算距离
    const scalar_t u = (points[n][0]+1)/2;
    const scalar_t v = (points[n][1]+1)/2;
    const scalar_t w = (points[n][2]+1)/2;

    //计算权重
    const scalar_t a = (1-v)*(1-w);
    const scalar_t b = (1-v)*w;
    const scalar_t c = v*(1-w);
    const scalar_t d = 1-a-b-c;

    feat_interp[n][f] = (1-u)*(a*feats[n][0][f] +
                            b*feats[n][1][f] +
                            c*feats[n][2][f] +
                            d*feats[n][3][f]) + 
                        u*(a*feats[n][4][f] +
                        b*feats[n][5][f] +
                        c*feats[n][6][f] +
                        d*feats[n][7][f]);
}

torch::Tensor trilinear_fw_cu(
    torch::Tensor feats,
    torch::Tensor points
){
    //先生成空的结果值，然后填充
    const int N = feats.size(0), F = feats.size(2);
    
    // 为每个kernal配置参数
    torch::Tensor feat_interp = torch::zeros({N, F}, feats.options());
    // feat_interp的加载设备、数据类型和feats一致

    const dim3 threads(16, 16); //每个thread包含一个(16, 16)的区域
    const dim3 blocks ((N+threads.x)/threads.x, (F+threads.y-1)/threads.y);
    // 密铺所有的thread

    //启动各个kernal
    AT_DISPATCH_FLOATING_TYPES(feats.type(), "trilinear_fw_cu",
    ([&] {
        trilinear_fw_kernel<scalar_t><<<blocks, threads>>>(
            feats.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            points.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            feat_interp.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return feat_interp;
}

torch::Tensor trilinear_bw_cu(
    const torch::Tensor dL_d_feat_interp,
    const torch::Tensor feats,
    const torch::Tensor points
){
    const int N = feats.size(0), F = feats.size(2);
    
    torch::Tensor dl_dfeats = torch::zeros({N, 8, F}, feats.options());

    const dim3 threads(16, 16); 
    const dim3 blocks ((N+threads.x)/threads.x, (F+threads.y-1)/threads.y);

    //启动各个kernal
    AT_DISPATCH_FLOATING_TYPES(feats.type(), "trilinear_fw_cu",
    ([&] {
        trilinear_bw_kernel<scalar_t><<<blocks, threads>>>(
            dL_d_feat_interp.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            feats.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            points.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            dl_dfeats.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return dl_dfeats;

}